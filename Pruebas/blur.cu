#include "hip/hip_runtime.h"
%%writefile Blur.cu
#include <stdio.h>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <omp.h>
#include <sys/time.h>

using namespace cv;

Mat *get_sharpen_image(Mat *, Mat *, Mat *, Mat *, int , Size  );
Mat *get_blur_image(Mat *, Mat *, int , Size );
Mat *get_high_pass_image(Mat *, Mat *, Mat *, int , Size );

void matToUchar(Mat, uchar *, int, int);
void ucharToMat(uchar *, Mat, int, int);

void matToUchar(Mat frame, uchar *uFrame, int width, int height)
{
    for (int ch = 0; ch < 3; ch++)
        for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++)
                uFrame[(ch * width * height) + (i * width + j)] = frame.at<Vec3b>(i, j)[ch];
               
}

// Function to cast uchar to Mat
void ucharToMat(uchar *uFrame, Mat frame, int width, int height)
{
    for (int ch = 0; ch < 3; ch++)
        for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++)
                frame.at<Vec3b>(i, j)[ch] = uFrame[(ch * width * height) + (i * width + j)];
}


__global__ void get_blur_image(uchar *d_original_image, uchar *d_blur_image, int width, int height, int nThreads){
    
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    // Position variables to get the optical flow
    int startPos = (thread_id < (width * height) % nThreads) ? ((width * height) / nThreads) * thread_id + thread_id : ((width * height) / nThreads) * thread_id + (width * height) % nThreads;
    int endPos = (thread_id < (width * height) % nThreads) ? startPos + ((width * height) / nThreads) : startPos + ((width * height) / nThreads) - 1; 
    int i = (startPos / width), j = (startPos % width);

    int filterBlur[25] = {1,4,7,4,1,4,16,26,16,4,7,26,41,26,7,4,16,26,16,4,1,4,7,4,1};

    for (; startPos <= endPos; startPos++){
        if(i <= 1 || j <= 1 || i >= height - 1 || j >= width - 1 ){

           //Blue
           *(d_blur_image + (i * width + j)) = (int) *(d_original_image + (i * width + j)) / 3;
           //Green
           *(d_blur_image + (width * height) + (i * width + j)) = (int) *(d_original_image + (i * width + j)) / 3;
           //Red
           *(d_blur_image + (2 * width * height) + (i * width + j)) = (int) *(d_original_image + (i * width + j)) / 3;

        }else{
              int positionFilter = 0;
              int blueBlur = 0;
              int greenBlur = 0;
              int redBlur = 0;
              for(int k = -12; k <= 12 ; k++){
                  blueBlur += *(d_original_image + (i * width + j) + k) * filterBlur[positionFilter];
                  greenBlur += *(d_original_image + (width * height) + (i * width + j) + k) * filterBlur[positionFilter];
                  redBlur += *(d_original_image + (2 * width * height) + (i * width + j) + k) * filterBlur[positionFilter];
                  positionFilter++;
              }
              *(d_blur_image + (i * width + j)) = (int) blueBlur / 273;
              *(d_blur_image + (width * height) + (i * width + j)) = (int) greenBlur / 273;
              *(d_blur_image + (2 * width * height) + (i * width + j)) = (int) redBlur / 273;
        }
        j += 1;
        if (j == width){
            i += 1;
            j = 0;
        }
     }
}


int main(int argc, char** argv )
{

    // Declare the variables for time measurement
    struct timeval tval_before, tval_after, tval_result;

    Mat image;
    //Mat *result_sharpen;
    Mat imageChannel[3];    
    Mat saveImage;
    Size frameSize;
    
    hipError_t err = hipSuccess;
    
    std::vector<Mat> mChannels;

    if ( argc != 2 ){
        printf("usage: test <Image_Path>\n");
        return -1;
    }
    // Get start time
    gettimeofday(&tval_before, NULL);

    image = imread(argv[1], 1);
    frameSize = image.size();

    if ( !image.data ){
        printf("No image data \n");
        return -1;
    }

    split(image, imageChannel);

    int width = frameSize.width;
    int height = frameSize.height;
    int channels = 3;
    
    int size = height * width * channels * sizeof(uchar);


    uchar *d_original_image, *d_blur_image;
    int nThreads = 64;
    /*Variable para el número de bloques*/
    int nBlocks = 40;

    uchar *originalImage = (uchar *)malloc(size);
    uchar *blurImage = (uchar *)malloc(size);

    matToUchar(image, originalImage, width, height);
    
    //Inputs
    err = hipMalloc((void **)&d_original_image, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device d_original_image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_original_image, originalImage, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy ker from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Outputs
    err = hipMalloc((void **)&d_blur_image, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device d_original_image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    get_blur_image<<<nBlocks, nThreads>>>(d_original_image, d_blur_image, width, height, nBlocks * nThreads);
    hipDeviceSynchronize();

    err = hipMemcpy(blurImage, d_blur_image, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy blurImage from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    Mat imageBlur = Mat::zeros(Size(width, height), CV_8UC3);

    ucharToMat(blurImage, imageBlur, width, height);

    // Calcular los tiempos en tval_result
    //  Get end time
    gettimeofday(&tval_after, NULL);

    timersub(&tval_after, &tval_before, &tval_result);
    /*Imprimir informe*/
    printf("------------------------------------------------------------------------------\n");
    printf("Tiempo de ejecución: %ld.%06ld s \n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);

    //mChannels = {imageBlur[0],imageBlur[1],imageBlur[2]};
    //merge(mChannels, saveImage);

    if(imwrite("blur.png", imageBlur) == false){
        std::cout << "Saving fail"<<std::endl;
        return -1;
    }

    waitKey(0);

    return 0;
}